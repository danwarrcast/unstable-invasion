
#include <hip/hip_runtime.h>
#include <iostream>
#include <fstream>
#include <sstream>
#include <string>
#include <math.h>
#include <hiprand/hiprand_kernel.h>
#include <algorithm>

bool to_bool(std::string str)
{
    std::transform(str.begin(), str.end(), str.begin(), ::tolower);
    std::istringstream is(str);
    bool b;
    is >> std::boolalpha >> b;
    return b;
}

__global__
void setup_kernel(hiprandState *state, int N, unsigned long SEED) 
{        
    int i, idx = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = gridDim.x * blockDim.x;

    // Init Random vector
    for (i = idx; i < N; i += stride)
    {
        hiprand_init(SEED + i, 0, 0, &state[i]);
    }

}

__device__
int winstrain(int a, int b, int c, double s1, double s3, double rannum)
{
    double g_a = 1.0 - s3*(double)(a-1)*(double)(a-2)/2.0 - s1*(double)(2-a)*(double)(3-a)/2.0;
    double g_b = 1.0 - s3*(double)(b-1)*(double)(b-2)/2.0 - s1*(double)(2-b)*(double)(3-b)/2.0;
    double g_c = 1.0 - s3*(double)(c-1)*(double)(c-2)/2.0 - s1*(double)(2-c)*(double)(3-c)/2.0;
  
    double vec[6] = {g_a/(g_a+g_b+g_c), double(a), g_b/(g_a+g_b+g_c), double(b), g_c/(g_a+g_b+g_c), double(c)};
  
    double G = 0.0;
    double result;
  
    for (int s = 0; s < 3; s++) {
      G += vec[2*s];
      if (G > rannum) {
        result = (int)vec[2*s + 1];
        break;
      }
    }
  
    return result;
}

__global__
void update_odd(int N, int *l_d, int *l_u, int L, int L2, double s1, double s3, double mu, hiprandState *state)
{
    hiprandState localState;
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;
    for (int i = index; i < N; i += stride)
    {
        int x = index % L;
        int y = index / L;

        int yp = y<L2-1?y+1:0;
        //int ym = y>0?y-1:L2-1; not used in odd updates
        int xp = x<L-1?x+1:0;
        int xm = x>0?x-1:L-1;

        if (y % 2 == 0)
        {
           int a = l_d[index];
           int b = l_d[yp * L + x];
           int c = l_d[yp * L + xp];
           if (a == b && a == c)
           {
                l_u[index] = a;
           } else {
               localState = state[i];
               l_u[index] = winstrain(a, b, c, s1, s3, hiprand_uniform(&localState));
               state[i] = localState;
           }
        } else {
           int a = l_d[index];
           int b = l_d[yp * L + x];
           int c = l_d[yp * L + xm];
           if (a == b && a == c)
           {
                l_u[index] = a;
           } else {
               localState = state[i];
               l_u[index] = winstrain(a, b, c, s1, s3, hiprand_uniform(&localState));
               state[i] = localState;
           }
        }
        //if mu is non-zero, check for mutation event
        if (l_u[index] == 2 && mu > 0.0000000001)
        {
            localState = state[i];
            if (hiprand_uniform(&localState) < mu) l_u[index] = 3;
            state[i] = localState;
        }
    }
}

__global__
void update_even(int N, int *l_d, int *l_u, int L, int L2, double s1, double s3, double mu, hiprandState *state)
{
    hiprandState localState;
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;
    for (int i = index; i < N; i += stride)
    {
        int x = index % L;
        int y = index / L;

        //int yp = y<L2-1?y+1:0; not used in even updates
        int ym = y>0?y-1:L2-1;
        int xp = x<L-1?x+1:0;
        int xm = x>0?x-1:L-1;

        if (y % 2 == 0)
        {
           int a = l_d[index];
           int b = l_d[ym * L + x];
           int c = l_d[ym * L + xp];
           if (a == b && a == c)
           {
                l_u[index] = a;
           } else {
               localState = state[i];
               l_u[index] = winstrain(a, b, c, s1, s3, hiprand_uniform(&localState));
               state[i] = localState;
           }
        } else {
           int a = l_d[index];
           int b = l_d[ym * L + x];
           int c = l_d[ym * L + xm];
           if (a == b && a == c)
           {
                l_u[index] = a;
           } else {
               localState = state[i];
               l_u[index] = winstrain(a, b, c, s1, s3, hiprand_uniform(&localState));
               state[i] = localState;
           }
        }
        //if mu is non-zero, check for mutation event
        if (l_u[index] == 2 && mu > 0.0000000001)
        {
            localState = state[i];
            if (hiprand_uniform(&localState) < mu) l_u[index] = 3;
            state[i] = localState;
        }
    }
}

int main(int argc, char* argv[])
{
    hipError_t cudaStatus;

    int lattsize, lattsize2, nogen, numruns;
    double mu, s1, s3;
    bool image;

    lattsize = atoi(argv[1]);
    lattsize2 = lattsize * 4;
    s3 = atof(argv[2]);
    s1 = s3 - atof(argv[3]);
    mu = atof(argv[4]);
    nogen = atoi(argv[5]);
    numruns = atoi(argv[6]);
    image = false;

    int N = lattsize * lattsize2;
    int *lattdown;
    int *lattup;
    hiprandState* devStates;

    int left = lattsize2 / 2 - lattsize / 4;
    int right = lattsize2 / 2 + lattsize / 4;

    int nogen2 = floor(log2(nogen));
    nogen = pow(2, nogen2);

    int blockSize = 256;
    int numBlocks = (N + blockSize - 1) / blockSize;

    if (argc == 8 && to_bool(argv[7]))
    {
      image = true;
    }
    else if (argc == 8 && !to_bool(argv[7]))
    {
      image = false;
    }

    std::string statsfilename;  
    std::ofstream outstats;
    std::ofstream outstats2;
    std::ofstream outstats3;
    std::ifstream testoutstats;
    int filecounter=0;
    std::string tempstr;
    std::ostringstream tempstring;	

    tempstring << filecounter;

    statsfilename += "diffusion_out/diffusion_run";
    statsfilename += tempstring.str();

    testoutstats.open(statsfilename.c_str());
    testoutstats.close();
	
    while (!testoutstats.fail())
    {
      tempstr = tempstring.str();
      statsfilename.erase(statsfilename.end()-tempstr.size(),statsfilename.end());
      filecounter++;
      tempstring.str("");
      tempstring.clear();
      tempstring << filecounter;
      statsfilename += tempstring.str();
      testoutstats.open(statsfilename.c_str());
      testoutstats.close();
    }

    testoutstats.clear(std::ios::failbit);
    outstats.open(statsfilename.c_str());

    std::cout << statsfilename.c_str() << std::endl;

    std::cout << "# Lx: " << lattsize << " Lt: " << nogen << " N_runs: " << numruns << " Has_Image: " << image << std::endl;
    std::cout << " # sW = " << s3 << " b = " << s3 - s1 << " mu = " << mu << std::endl;

    outstats << "# Lx: " << lattsize << " Lt: " << nogen << " N_runs: " << numruns << " Has_Image: " << image << std::endl;
    outstats << " # sW = " << s3 << " b = " << s3 - s1 << " mu = " << mu << std::endl;

    cudaStatus = hipMallocManaged(&lattdown, N * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }
    cudaStatus = hipMallocManaged(&lattup, N * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }
    cudaStatus = hipMalloc(&devStates, N * sizeof(hiprandState));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    setup_kernel <<<numBlocks, blockSize>>> (devStates, N, time(0));
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching setup_kernel!\n", cudaStatus);
        goto Error;
    }

    for (int q = 0; q < numruns; q++)
    {
        //initialize lattice
        for (int i = 0; i < N; i++)
        {
            lattdown[i] = 1;
            lattup[i] = 0;
            int j = i / lattsize;
            if (j > left && j <= right) lattdown[i] = 2;
        }

        int t2 = 0;
        int count = 1;
        bool pow_2 = false;
        for (int t = 1; t < nogen; t++)
        {
            if (t == count) {
                pow_2 = true;
                if (t > 1) t2++;
                count = count * 2;
            } else {
                pow_2 = false;
            }

            //even step
            update_odd <<<numBlocks, blockSize>>> (N, lattdown, lattup, lattsize, lattsize2, s1, s3, mu, devStates);
            cudaStatus = hipDeviceSynchronize();
            if (cudaStatus != hipSuccess) {
                fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching update_odd!\n", cudaStatus);
                goto Error;
            }

            //if (pow_2) find_width();

            ++t;
            if (t == count) {
                pow_2 = true;
                if (t > 1) t2++;
                count = count * 2;
            } else {
                pow_2 = false;
            }

            //odd step
            update_even <<<numBlocks, blockSize>>> (N, lattup, lattdown, lattsize, lattsize2, s1, s3, mu, devStates);
            cudaStatus = hipDeviceSynchronize();
            if (cudaStatus != hipSuccess) {
                fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching update_even!\n", cudaStatus);
                goto Error;
            }

            //if (pow_2) find_width();
        }
    }

    int *lattup_host;
    lattup_host = new int[N];
    hipMemcpy(lattup_host, lattup, N * sizeof(int), hipMemcpyDeviceToHost);

    if (image)
    {
        for (int i = 0; i < N; i++)
        {
            if (lattup_host[i] == 1) continue;
            int x = i % lattsize;
            int y = i / lattsize;
            double xx = (double)x - 0.5 * (y % 2);
            double yy = (double)y * sqrt(3.0)/2.0;
            outstats << xx << "," << yy << "," << lattup_host[i] << "\n";
        }
        outstats << std::endl;
    }
    
    outstats.close();

    hipFree(lattdown);
    hipFree(lattup);

    return 0;

    Error:
        hipFree(lattdown);
        hipFree(lattup);
        hipFree(devStates);
        return (int)cudaStatus;
        
}